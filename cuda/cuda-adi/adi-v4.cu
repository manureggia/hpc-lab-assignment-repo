#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "adi.h"


#define gpuErrchk(ans)                  \
{                                       \
	gpuAssert((ans), __FILE__, __LINE__); \
}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

void print_array(int n, DATA_TYPE *X)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%0.2f", X[i * n + j]);
			if ((i * n + j) % n == 0)
				printf("\n");
		}
	}
	printf("\n");
}

// Confronta due matrici per verificare la correttezza
int compare_matrices(DATA_TYPE *X_host, DATA_TYPE *X_copyice, int n)
{
  int return_value = 1;
  for (int i = 0; i < n; i++)
  {
    for (int j = 0; j < n; j++)
    {
      if (fabs(X_host[i * n + j] - X_copyice[i * n + j]) > 1e-6)
      {
        printf("Mismatch at (%d, %d): Host = %f, Device = %f\n", i, j, X_host[i * n + j], X_copyice[i * n + j]);
        return_value = 0;
      }
    }
  }
  return return_value;
}

// Kernel host
void kernel_adi_host(int tsteps, int n, DATA_TYPE *X, const DATA_TYPE *A, DATA_TYPE *B)
{
	/**
	 * Questo codice implementa una risoluzione dell'algoritmo Alternating-Direction Implicit (ADI) 
	 * su una griglia bidimensionale. È strutturato per aggiornare le soluzioni delle equazioni 
	 * differenziali in due passaggi: uno lungo le colonne e uno lungo le righe della griglia
	 */

	/** Ad ogni iterazione, il sistema viene aggiornato lungo le colonne e le righe */
	for (int t = 0; t < tsteps; t++)
	{
		/**
		 * Aggiornamento lungo le colonne:
		 * Per ogni riga:
		 * 1. si applica l'eliminazione in avanti per ridurre il sistema tridiagonale lungo le colonne.
		 * 2. X rappresenta il vettore delle soluzioni.
		 * 3. A e B rappresentano i coefficienti delle equazioni differenziali.
		 */
		for (int row = 0; row < n; row++)
		{
			for (int col = 1; col < n; col++)
			{
				X[row * n + col] -= X[row * n + col - 1] * A[row * n + col] / B[row * n + col - 1];
				B[row * n + col] -= A[row * n + col] * A[row * n + col] / B[row * n + col - 1];
			}
		}

		/**
		 * Normalizzazione:
		 * Il valore della soluzione nella parte inferiore della colonna è normalizzato dividendo 
		 * per il coefficiente B
		 */
		for (int col = 0; col < n; col++)
			X[col * n + (n - 1)] /= B[col * n + (n - 1)];

		
		/**
		 * Sostituzione all'indietro (Back Substitution):
		 * Dopo l'eliminazione in avanti, si risolvono i valori risalendo lungo la colonna
		 */
		for (int row = 0; row < n; row++)
			for (int col = 0; col < n - 2; col++)
				X[row * n + (n - col - 2)] = (X[row * n + (n - col - 2)] - X[row * n + (n - col - 3)] * A[row * n + (n - col - 3)]) / B[row * n + (n - col - 3)];
		

		/**
		 * Aggiornamento lungo le righe:
		 * Qui il processo si applica lungo le righe.
		 * Stesso approccio dell'eliminazione lungo le colonne, ma con iterazione spaziale lungo i1.
		 */
		for (int row = 1; row < n; row++)
		{
			for (int col = 0; col < n; col++)
			{
				X[row * n + col] -= X[(row - 1) * n + col] * A[row * n + col] / B[(row - 1) * n + col];
				B[row * n + col] -= A[row * n + col] * A[row * n + col] / B[(row - 1) * n + col];
			}
		}

		/**
		 * Normalizzazione:
		 * Si normalizza l'ultima riga dividendo per B.
		 */
		for (int col = 0; col < n; col++)
			X[(n - 1) * n + col] /= B[(n - 1) * n + col];

		/**
		 * Back-substitution:
		 * Anche in questo caso, si risolve il sistema risalendo lungo le righe
		 */
		for (int row = 0; row < n - 2; row++)
			for (int col = 0; col < n; col++)
				X[(n - 2 - row) * n + col] = (X[(n - 2 - row) * n + col] - X[(n - row - 3) * n + col] * A[(n - 3 - row) * n + col]) / B[(n - 2 - row) * n + col];
	}
}

__global__ void kernel_column_forward_elimination(int n, DATA_TYPE *X, const DATA_TYPE *A, DATA_TYPE *B)
{
	// SENZA SHARED MEMORY
	// -----------------------------------------------
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n)
	{
		for (int col = 1; col < n; col++) 
		{
			int idx = row * n + col;
			int prev_idx = row * n + (col - 1);
			X[idx] -= X[prev_idx] * A[idx] / B[prev_idx];
			B[idx] -= A[idx] * A[idx] / B[prev_idx];
		}
	}

	/**
	 * Gli elementi di X, A, e B sono utilizzati più volte all'interno dello stesso blocco di thread, 
	 * possono essere caricati nella shared memory. La shared memory riduce il numero di accessi alla memoria globale, 
	 * minimizzando il tempo di latenza.
	 */

	// todo ...
}
__global__ void kernel_column_norm(int n, DATA_TYPE *X, const DATA_TYPE *B)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n) 
	{
		int last_col_idx = row * n + (n - 1);
		X[last_col_idx] /= B[last_col_idx];
	}
}
__global__ void kernel_column_back_sostitution(int n, DATA_TYPE *X, const DATA_TYPE *A, const DATA_TYPE *B)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n) 
	{
		for (int col = n - 2; col >= 0; col--) 
		{
			int idx = row * n + col;
			int next_idx = row * n + (col + 1);
			X[idx] = (X[idx] - X[next_idx] * A[next_idx]) / B[idx];
		}
	}
}
__global__ void kernel_row_forward_elimination(int n, DATA_TYPE *X, const DATA_TYPE *A, DATA_TYPE *B)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < n)
	{
		for (int row = 1; row < n; row++) 
		{
			int idx = row * n + col;
			int prev_idx = (row - 1) * n + col;
			X[idx] -= X[prev_idx] * A[idx] / B[prev_idx];
			B[idx] -= A[idx] * A[idx] / B[prev_idx];
		}
	}
}
__global__ void kernel_row_norm(int n, DATA_TYPE *X, const DATA_TYPE *B)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < n)
	{
		int last_row_idx = (n - 1) * n + col;
		X[last_row_idx] /= B[last_row_idx];
	}
}
__global__ void kernel_row_back_sostitution(int n, DATA_TYPE *X, const DATA_TYPE *A, const DATA_TYPE *B)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if (col < n)
	{
		for (int row = n - 2; row >= 0; row--) 
		{
			int idx = row * n + col;
			int next_idx = (row + 1) * n + col;
			X[idx] = (X[idx] - X[next_idx] * A[next_idx]) / B[idx];
		}
	}
}

int main()
{
	const int n = N;
	const int tsteps = TSTEPS;
	const int bytes = sizeof(DATA_TYPE) * n * n;
	struct timespec rt[2];

	// Lato GPU sono necessari i seguenti dati:
	// - X[] lettura/scrittura
	// - B[] lettura/scrittura
	// - A[] solo lettura
	// Quindi possiamo usare la memoria unificata per queste 3 variabili
	// X=d_X
	// B=d_B
	// A=d_A

	DATA_TYPE *X, *A, *B;
	gpuErrchk(hipMallocManaged(&A, bytes));
	gpuErrchk(hipMallocManaged(&X, bytes));
	gpuErrchk(hipMallocManaged(&B, bytes));

	DATA_TYPE* X_copy = (DATA_TYPE*)malloc(bytes);
	DATA_TYPE* B_copy = (DATA_TYPE*)malloc(bytes);

	#pragma omp parallel
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			int idx 			= i * n + j;
			X[idx] 				= ((DATA_TYPE)i * (j + 1) + 1) / n;
			A[idx] 				= ((DATA_TYPE)i * (j + 2) + 2) / n;
			B[idx] 				= ((DATA_TYPE)i * (j + 3) + 3) / n;
			X_copy[idx] 	= ((DATA_TYPE)i * (j + 1) + 1) / n;
			B_copy[idx] 	= ((DATA_TYPE)i * (j + 3) + 3) / n;
		}
	}

	// call ADI on host
	{
		clock_gettime(CLOCK_REALTIME, rt);
		kernel_adi_host(tsteps, n, X_copy, A, B_copy);
		clock_gettime(CLOCK_REALTIME, rt + 1);

		double wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
		printf("ADI (Host): %9.3f sec\n", wt);
	}

	// call ADI on GPU
	{
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

		clock_gettime(CLOCK_REALTIME, rt);
		/**
		 * Come abbiamo visto nell'implementazione lato host, l'algoritmo adi si compone di diverse
		 * passi fondamentali:
		 * [1] Aggiornamento lungo le colonne: 
		 * 		[1.1] eliminazione in avanti (Forward Elimination)
		 * 		[1.2] normalizzazione
		 * 		[1.3] sostituzione all'indietro (Back Substitution)
		 * [2] Aggiornamento lungo le righe:
		 * 		[2.1] eliminazione in avanti
		 * 		[2.2] normalizzazione
		 * 		[2.3] sostituzione all'indietro
		 * 
		 * Nell'algoritmo ADI, alcune operazioni possono essere eseguite in parallelo perché non dipendono 
		 * direttamente dai risultati degli altri calcoli per ogni iterazione spaziale.
		 * Le operazioni possono essere parallelizzate per righe durante l'aggiornamento lungo le colonne e 
		 * per colonne durante l'aggiornamento lungo le righe.
		 */
		for (int t = 0; t < tsteps; t++)
		{
			// ------------------------------------------------
			// [1] Aggiornamento lungo le colonne
			// ------------------------------------------------
			// [1.1] eliminazione in avanti (Forward Elimination): 
			// gli aggiornamenti lungo una colonna di una stessa riga dipendono dal valore precedente
			// nella stessa riga, quindi non è parallelizzabile lungo le colonne,
			// ma l'operazione per righe differenti è indipendente.
			kernel_column_forward_elimination<<<grid, block>>>(n, X, A, B);
			gpuErrchk(hipPeekAtLastError());  
			gpuErrchk(hipDeviceSynchronize());
			// [1.2] normalizzazione: 
			// parallelizzabile per riga; ogni riga è indipendente.
			kernel_column_norm<<<grid, block>>>(n, X, B);
			gpuErrchk(hipPeekAtLastError());  
			gpuErrchk(hipDeviceSynchronize());
			// [1.3] sostituzione all'indietro (Back Substitution):
			// parallelizzabile per riga; anche qui, ogni riga rappresenta un sistema tridiagonale indipendente.
			// L'operazione lungo colonne dipende dai valori precedenti della stessa riga.
			kernel_column_back_sostitution<<<grid, block>>>(n, X, A, B);
			gpuErrchk(hipPeekAtLastError());  
			gpuErrchk(hipDeviceSynchronize());

			// ------------------------------------------------
			// [2] Aggiornamento lungo le righe
			// ------------------------------------------------
			// [2.1] eliminazione in avanti:
			// parallelizzabile per colonna; ogni colonna della griglia rappresenta un sistema 
			// tridiagonale indipendente.
			// Gli aggiornamenti lungo una riga dipendono dal valore precedente nella stessa colonna, 
			// quindi non è parallelizzabile lungo le righe, ma può essere parallelo tra colonne diverse.
			kernel_row_forward_elimination<<<grid, block>>>(n, X, A, B);
			gpuErrchk(hipPeekAtLastError());  
			gpuErrchk(hipDeviceSynchronize());
			// [2.2] normalizzazione: 
			// parallelizzabile per colonna; ogni colonna è indipendente.
			kernel_row_norm<<<grid, block>>>(n, X, B);
			gpuErrchk(hipPeekAtLastError());  
			gpuErrchk(hipDeviceSynchronize());
			// [2.3] sostituzione all'indietro:
			// parallelizzabile per colonna; simile all'eliminazione in avanti, 
			// ogni colonna rappresenta un sistema tridiagonale indipendente.
			kernel_row_back_sostitution<<<grid, block>>>(n, X, A, B);
			gpuErrchk(hipPeekAtLastError());  
			gpuErrchk(hipDeviceSynchronize());
		}    
		
		clock_gettime(CLOCK_REALTIME, rt + 1);

		double wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
		printf("ADI (GPU): %9.3f sec\n", wt);
	}


	if (compare_matrices(X, X_copy, n))
	{
		printf("Risultati Host e Device CORRETTI!\n");
	}
	else
	{
		printf("Risultati Host e Device NON corrispondono!\n");
	}

	// Liberazione memoria

	free(X_copy);
	free(B_copy);
	gpuErrchk(hipFree(X));
	gpuErrchk(hipFree(A));
	gpuErrchk(hipFree(B));
	return 0;
}
