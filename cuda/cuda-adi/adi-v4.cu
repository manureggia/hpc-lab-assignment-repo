#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <omp.h>

#include "adi.h"


#define gpuErrchk(ans)                  \
{                                       \
	gpuAssert((ans), __FILE__, __LINE__); \
}
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort)
			exit(code);
	}
}

void init_array(int n, DATA_TYPE *X, DATA_TYPE *X_DEV, DATA_TYPE *A, DATA_TYPE *B, DATA_TYPE *B_DEV)
{
	#pragma omp parallel
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			X[i * n + j] 			= ((DATA_TYPE)i * (j + 1) + 1) / n;
			X_DEV[i * n + j] 	= ((DATA_TYPE)i * (j + 1) + 1) / n;
			A[i * n + j] 			= ((DATA_TYPE)i * (j + 2) + 2) / n;
			B[i * n + j] 			= ((DATA_TYPE)i * (j + 3) + 3) / n;
			B_DEV[i * n + j] 	= ((DATA_TYPE)i * (j + 3) + 3) / n;
		}
	}
}

void print_array(int n, DATA_TYPE *X)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
		{
			printf("%0.2f", X[i * n + j]);
			if ((i * n + j) % n == 0)
				printf("\n");
		}
	}
	printf("\n");
}

// Confronta due matrici per verificare la correttezza
int compare_matrices(DATA_TYPE *X_host, DATA_TYPE *X_device, int n)
{
  int return_value = 1;
  for (int i = 0; i < n; i++)
  {
    for (int j = 0; j < n; j++)
    {
      if (fabs(X_host[i * n + j] - X_device[i * n + j]) > 1e-6)
      {
        printf("Mismatch at (%d, %d): Host = %f, Device = %f\n", i, j, X_host[i * n + j], X_device[i * n + j]);
        return_value = 0;
      }
    }
  }
  return return_value;
}

// Kernel host
void kernel_adi_host(int tsteps, int n, DATA_TYPE *X, DATA_TYPE *A, DATA_TYPE *B)
{
	/**
	 * Questo codice implementa una risoluzione dell'algoritmo Alternating-Direction Implicit (ADI) 
	 * su una griglia bidimensionale. È strutturato per aggiornare le soluzioni delle equazioni 
	 * differenziali in due passaggi: uno lungo le colonne e uno lungo le righe della griglia
	 */

	/** Ad ogni iterazione, il sistema viene aggiornato lungo le colonne e le righe */
	for (int t = 0; t < tsteps; t++)
	{
		/**
		 * Aggiornamento lungo le colonne:
		 * Per ogni riga:
		 * 1. si applica l'eliminazione in avanti per ridurre il sistema tridiagonale lungo le colonne.
		 * 2. X rappresenta il vettore delle soluzioni.
		 * 3. A e B rappresentano i coefficienti delle equazioni differenziali.
		 */
		for (int row = 0; row < n; row++)
		{
			for (int col = 1; col < n; col++)
			{
				X[row * n + col] -= X[row * n + col - 1] * A[row * n + col] / B[row * n + col - 1];
				B[row * n + col] -= A[row * n + col] * A[row * n + col] / B[row * n + col - 1];
			}
		}

		/**
		 * Normalizzazione:
		 * Il valore della soluzione nella parte inferiore della colonna è normalizzato dividendo 
		 * per il coefficiente B
		 */
		for (int col = 0; col < n; col++)
			X[col * n + (n - 1)] /= B[col * n + (n - 1)];

		
		/**
		 * Sostituzione all'indietro (Back Substitution):
		 * Dopo l'eliminazione in avanti, si risolvono i valori risalendo lungo la colonna
		 */
		for (int row = 0; row < n; row++)
			for (int col = 0; col < n - 2; col++)
				X[row * n + (n - col - 2)] = (X[row * n + (n - col - 2)] - X[row * n + (n - col - 3)] * A[row * n + (n - col - 3)]) / B[row * n + (n - col - 3)];
		

		/**
		 * Aggiornamento lungo le righe:
		 * Qui il processo si applica lungo le righe.
		 * Stesso approccio dell'eliminazione lungo le colonne, ma con iterazione spaziale lungo i1.
		 */
		for (int row = 1; row < n; row++)
		{
			for (int col = 0; col < n; col++)
			{
				X[row * n + col] -= X[(row - 1) * n + col] * A[row * n + col] / B[(row - 1) * n + col];
				B[row * n + col] -= A[row * n + col] * A[row * n + col] / B[(row - 1) * n + col];
			}
		}

		/**
		 * Normalizzazione:
		 * Si normalizza l'ultima riga dividendo per B.
		 */
		for (int col = 0; col < n; col++)
			X[(n - 1) * n + col] /= B[(n - 1) * n + col];

		/**
		 * Back-substitution:
		 * Anche in questo caso, si risolve il sistema risalendo lungo le righe
		 */
		for (int row = 0; row < n - 2; row++)
			for (int col = 0; col < n; col++)
				X[(n - 2 - row) * n + col] = (X[(n - 2 - row) * n + col] - X[(n - row - 3) * n + col] * A[(n - 3 - row) * n + col]) / B[(n - 2 - row) * n + col];
	}
}


int main()
{
	const int n = N;
	const int tsteps = TSTEPS;
	const int bytes = sizeof(DATA_TYPE) * n * n;
	struct timespec rt[2];

	DATA_TYPE* X 			= (DATA_TYPE*)malloc(bytes);
	DATA_TYPE* X_dev 	= (DATA_TYPE*)malloc(bytes);
	DATA_TYPE* A 			= (DATA_TYPE*)malloc(bytes);
	DATA_TYPE* B 			= (DATA_TYPE*)malloc(bytes);
	DATA_TYPE* B_dev 	= (DATA_TYPE*)malloc(bytes);
	init_array(n, X, X_dev, A, B, B_dev);

	// call ADI on host
	{
		clock_gettime(CLOCK_REALTIME, rt);
		kernel_adi_host(tsteps, n, X, A, B);
		clock_gettime(CLOCK_REALTIME, rt + 1);

		double wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
		printf("ADI (Host): %9.3f sec\n", wt);
	}

	// Allocazione memoria GPU
	DATA_TYPE *d_X, *d_A, *d_B;
	hipMalloc(&d_X, bytes);
	hipMalloc(&d_A, bytes);
	hipMalloc(&d_B, bytes);
	hipMemcpy(d_X, X_dev, 	bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_A, A, 			bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B_dev, 	bytes, hipMemcpyHostToDevice);

	// call ADI on GPU
	{
    const dim3 block(BLOCK_SIZE);
    const dim3 grid((n + BLOCK_SIZE - 1) / BLOCK_SIZE);

		clock_gettime(CLOCK_REALTIME, rt);
		/**
		 * Come abbiamo visto nell'implementazione lato host, l'algoritmo adi si compone di diverse
		 * passi fondamentali:
		 * [1] Aggiornamento lungo le colonne: 
		 * 		[1.1] eliminazione in avanti (Forward Elimination)
		 * 		[1.2] normalizzazione
		 * 		[1.3] sostituzione all'indietro (Back Substitution)
		 * [2] Aggiornamento lungo le righe:
		 * 		[2.1] eliminazione in avanti
		 * 		[2.2] normalizzazione
		 * 		[2.3] sostituzione all'indietro
		 * 
		 * Nell'algoritmo ADI, alcune operazioni possono essere eseguite in parallelo perché non dipendono 
		 * direttamente dai risultati degli altri calcoli per ogni iterazione spaziale.
		 * Le operazioni possono essere parallelizzate per righe durante l'aggiornamento lungo le colonne e 
		 * per colonne durante l'aggiornamento lungo le righe.
		 */
		for (int t = 0; t < tsteps; t++)
		{
			// ------------------------------------------------
			// [1] Aggiornamento lungo le colonne
			// ------------------------------------------------
			// [1.1] eliminazione in avanti (Forward Elimination): 
			// gli aggiornamenti lungo una colonna di una stessa riga dipendono dal valore precedente
			// nella stessa riga, quindi non è parallelizzabile lungo le colonne,
			// ma l'operazione per righe differenti è indipendente.
			kernel_column_forward_elimination<<<grid, block>>>(...);
			hipDeviceSynchronize();
			// [1.2] normalizzazione: 
			// parallelizzabile per riga; ogni riga è indipendente.
			kernel_column_norm<<<grid, block>>>(...);
			hipDeviceSynchronize();
			// [1.3] sostituzione all'indietro (Back Substitution):
			// parallelizzabile per riga; anche qui, ogni riga rappresenta un sistema tridiagonale indipendente.
			// L'operazione lungo colonne dipende dai valori precedenti della stessa riga.
			kernel_column_back_sostitution<<<grid, block>>>(...);
			hipDeviceSynchronize();

			// ------------------------------------------------
			// [2] Aggiornamento lungo le righe
			// ------------------------------------------------
			// [2.1] eliminazione in avanti:
			// parallelizzabile per colonna; ogni colonna della griglia rappresenta un sistema 
			// tridiagonale indipendente.
			// Gli aggiornamenti lungo una riga dipendono dal valore precedente nella stessa colonna, 
			// quindi non è parallelizzabile lungo le righe, ma può essere parallelo tra colonne diverse.
			kernel_row_forward_elimination<<<grid, block>>>(...);
			hipDeviceSynchronize();
			// [2.2] normalizzazione: 
			// parallelizzabile per colonna; ogni colonna è indipendente.
			kernel_row_norm<<<grid, block>>>(...);
			hipDeviceSynchronize();
			// [2.3] sostituzione all'indietro:
			// parallelizzabile per colonna; simile all'eliminazione in avanti, 
			// ogni colonna rappresenta un sistema tridiagonale indipendente.
			kernel_row_back_sostitution<<<grid, block>>>(...);
			hipDeviceSynchronize();
		}    
    hipMemcpy(X_dev, d_X, bytes, hipMemcpyDeviceToHost);
		gpuErrchk(hipPeekAtLastError());  
	
		clock_gettime(CLOCK_REALTIME, rt + 1);

		double wt = (rt[1].tv_sec - rt[0].tv_sec) + 1.0e-9 * (rt[1].tv_nsec - rt[0].tv_nsec);
		printf("ADI (GPU): %9.3f sec\n", wt);
	}


	if (compare_matrices(X, X_dev, n))
	{
		printf("Risultati Host e Device CORRETTI!\n");
	}
	else
	{
		printf("Risultati Host e Device NON corrispondono!\n");
	}

	// Liberazione memoria
	free(X);
	free(X_dev);
	free(A);
	free(B_dev);
	free(B);
	hipFree(d_X);
	hipFree(d_A);
	hipFree(d_B);
	return 0;
}
